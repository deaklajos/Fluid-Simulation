#include "TextureSurface3D.cuh"

TextureSurface3D::TextureSurface3D(const hipChannelFormatDesc& chanelDescriptor, size_t size)
{
	checkCudaErrors(hipMalloc3DArray(&dataArray, &chanelDescriptor, make_hipExtent(size, size, size), hipArraySurfaceLoadStore));

	hipResourceDesc resourceDescriptor;
	memset(&resourceDescriptor, 0, sizeof(hipResourceDesc));
	resourceDescriptor.resType = hipResourceTypeArray;
	resourceDescriptor.res.array.array = dataArray;

	hipTextureDesc textureDescriptor;
	memset(&textureDescriptor, 0, sizeof(hipTextureDesc));
	textureDescriptor.normalizedCoords = false; // access with unnormalized texture coordinates
	textureDescriptor.filterMode = hipFilterModeLinear; // linear interpolation
	// wrap texture coordinates
	textureDescriptor.addressMode[0] = hipAddressModeWrap;
	textureDescriptor.addressMode[1] = hipAddressModeWrap;
	textureDescriptor.addressMode[2] = hipAddressModeWrap;
	textureDescriptor.readMode = hipReadModeElementType;

	checkCudaErrors(hipCreateTextureObject(&texture, &resourceDescriptor, &textureDescriptor, NULL));

	checkCudaErrors(hipCreateSurfaceObject(&surface, &resourceDescriptor));
}

TextureSurface3D::~TextureSurface3D()
{
	checkCudaErrors(hipDestroyTextureObject(texture));
	checkCudaErrors(hipDestroySurfaceObject(surface));
	checkCudaErrors(hipFreeArray(dataArray));
}

hipTextureObject_t TextureSurface3D::getTexture() const
{
	return texture;
}

hipSurfaceObject_t TextureSurface3D::getSurface() const
{
	return surface;
}

hipArray* TextureSurface3D::getArray() const
{
	return dataArray;
}
